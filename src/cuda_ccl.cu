#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <fstream>
#include <chrono>
#include <vector>
#include <numeric>


#define BLOCK_ROWS 16
#define BLOCK_COLS 16


__global__ void Init_Labeling(unsigned int* labels, unsigned int* b_labels, const int label_rows, const int label_cols) {
    unsigned int row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
    unsigned int col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
    unsigned int labels_index = row * label_cols + col;

    if (row < label_rows && col < label_cols) {
        labels[labels_index] = labels_index;
        b_labels[labels_index] = labels_index;
    } 
}

__device__ unsigned int Find(unsigned int* labels, unsigned int label) {
    while (labels[label] != label) {
        label = labels[label];
    }
    return label;
}

__device__ void Union(unsigned int* labels, unsigned int a_label_idx, unsigned int b_label_idx) {
    bool end = false;
    do {
        unsigned int a_label = Find(labels, a_label_idx);
        unsigned int b_label = Find(labels, b_label_idx);

        if (a_label < b_label) {
            unsigned int old = atomicMin(&labels[b_label], a_label);
            end = (old == b_label);
            b_label_idx = old;
        } else if (b_label < a_label) {
            unsigned int old = atomicMin(&labels[a_label], b_label);
            end = (old == a_label);
            a_label_idx = old;
        } else {
            end = true;
        }
    } while (!end);
}

__global__ void Merge(unsigned int* img, unsigned int* labels, unsigned int* b_labels,  const int img_rows, const int img_cols) {
    const int label_rows = img_rows;
    const int label_cols = img_cols;
    unsigned int row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
    unsigned int col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
    unsigned int img_index = row * img_cols + col;
    unsigned int labels_index = row * label_cols + col;
    unsigned int b_labels_index = row * label_cols + col;

    if (row < label_rows && col < label_cols) {
        // Foreground pixels
#define CONDITION_B col>0 && row>1 && img[img_index - 2 * img_cols - 1] > 0
#define CONDITION_C row>1 && img[img_index - 2 * img_cols] > 0
#define CONDITION_D col+1<img_cols && row>1 && img[img_index - 2 * img_cols + 1] > 0
#define CONDITION_E col+2<img_cols && row>1 && img[img_index - 2 * img_cols + 2] > 0

#define CONDITION_G col>1 && row>0 && img[img_index - img_cols - 2] > 0
#define CONDITION_H col>0 && row>0 && img[img_index - img_cols - 1] > 0
#define CONDITION_I row>0 && img[img_index - img_cols] > 0
#define CONDITION_J col+1<img_cols && row>0 && img[img_index - img_cols + 1] > 0
#define CONDITION_K col+2<img_cols && row>0 && img[img_index - img_cols + 2] > 0

#define CONDITION_M col>1 && img[img_index - 2] > 0
#define CONDITION_N col>0 && img[img_index - 1] > 0
#define CONDITION_O img[img_index] > 0
#define CONDITION_P col+1<img_cols && img[img_index + 1] > 0

#define CONDITION_R col>0 && row+1<img_rows && img[img_index + img_cols - 1] > 0
#define CONDITION_S row+1<img_rows && img[img_index + img_cols] > 0
#define CONDITION_T col+1<img_cols && row+1<img_rows && img[img_index + img_cols + 1] > 0

// Action 1: No action
#define ACTION_1  
//			// Action 2: New label (the block has foreground pixels and is not connected to anything else)
#define ACTION_2  
            //Action P: Merge with block P
#define ACTION_3 Union(labels, labels_index, labels_index - 2 * img_cols - 2); 
            // Action Q: Merge with block Q
#define ACTION_4 Union(labels, labels_index, labels_index - 2 * img_cols);	
            // Action R: Merge with block R
#define ACTION_5 Union(labels, labels_index, labels_index - 2 * img_cols + 2); 
            // Action S: Merge with block S
#define ACTION_6 Union(labels, labels_index, labels_index - 2);  
            // Action 7: Merge labels of block P and Q
#define ACTION_7 Union(labels, labels_index, labels_index - 2 * img_cols - 2); \
            Union(labels, labels_index, labels_index - 2 * img_cols);			
            //Action 8: Merge labels of block P and R
#define ACTION_8 Union(labels, labels_index, labels_index - 2 * img_cols - 2); \
            Union(labels, labels_index, labels_index - 2 * img_cols + 2);			
            // Action 9 Merge labels of block P and S
#define ACTION_9 Union(labels, labels_index, labels_index - 2 * img_cols - 2); \
            Union(labels, labels_index, labels_index - 2);			
            // Action 10 Merge labels of block Q and R
#define ACTION_10 Union(labels, labels_index, labels_index - 2 * img_cols); \
            Union(labels, labels_index, labels_index - 2 * img_cols + 2);			
            // Action 11: Merge labels of block Q and S
#define ACTION_11 Union(labels, labels_index, labels_index - 2 * img_cols); \
            Union(labels, labels_index, labels_index - 2);			
            // Action 12: Merge labels of block R and S
#define ACTION_12 Union(labels, labels_index, labels_index - 2 * img_cols + 2); \
            Union(labels, labels_index, labels_index - 2);			
            // Action 13: not used
#define ACTION_13 
            // Action 14: Merge labels of block P, Q and S
#define ACTION_14 Union(labels, labels_index, labels_index - 2 * img_cols - 2); \
            Union(labels, labels_index, labels_index - 2 * img_cols); \
            Union(labels, labels_index, labels_index - 2);		
            //Action 15: Merge labels of block P, R and S
#define ACTION_15 Union(labels, labels_index, labels_index - 2 * img_cols - 2); \
            Union(labels, labels_index, labels_index - 2 * img_cols + 2); \
            Union(labels, labels_index, labels_index - 2);			
            //Action 16: labels of block Q, R and S
#define ACTION_16 Union(labels, labels_index, labels_index - 2 * img_cols); \
            Union(labels, labels_index, labels_index - 2 * img_cols + 2); \
            Union(labels, labels_index, labels_index - 2);	

        #include "drag_fg.inc.cuh"

#undef ACTION_1
#undef ACTION_2
#undef ACTION_3
#undef ACTION_4
#undef ACTION_5
#undef ACTION_6
#undef ACTION_7
#undef ACTION_8
#undef ACTION_9
#undef ACTION_10
#undef ACTION_11
#undef ACTION_12
#undef ACTION_13
#undef ACTION_14
#undef ACTION_15
#undef ACTION_16


#undef CONDITION_B
#undef CONDITION_C
#undef CONDITION_D
#undef CONDITION_E

#undef CONDITION_G
#undef CONDITION_H
#undef CONDITION_I
#undef CONDITION_J
#undef CONDITION_K

#undef CONDITION_M
#undef CONDITION_N
#undef CONDITION_O
#undef CONDITION_P

#undef CONDITION_R
#undef CONDITION_S
#undef CONDITION_T
    
    // Background pixels
#define CONDITION_B col>0 && row>1 && img[img_index - 2 * img_cols - 1] == 0
#define CONDITION_C row>1 && img[img_index - 2 * img_cols] == 0
#define CONDITION_D col+1<img_cols && row>1 && img[img_index - 2 * img_cols + 1] == 0
#define CONDITION_E col+2<img_cols && row>1 && img[img_index - 2 * img_cols + 2] == 0

#define CONDITION_G col>1 && row>0 && img[img_index - img_cols - 2] == 0
#define CONDITION_H col>0 && row>0 && img[img_index - img_cols - 1] == 0
#define CONDITION_I row>0 && img[img_index - img_cols] == 0
#define CONDITION_J col+1<img_cols && row>0 && img[img_index - img_cols + 1] == 0
#define CONDITION_K col+2<img_cols && row>0 && img[img_index - img_cols + 2] == 0

#define CONDITION_M col>1 && img[img_index - 2] == 0
#define CONDITION_N col>0 && img[img_index - 1] == 0
#define CONDITION_O img[img_index] == 0
#define CONDITION_P col+1<img_cols && img[img_index + 1] == 0

#define CONDITION_R col>0 && row+1<img_rows && img[img_index + img_cols - 1] == 0
#define CONDITION_S row+1<img_rows && img[img_index + img_cols] == 0
#define CONDITION_T col+1<img_cols && row+1<img_rows && img[img_index + img_cols + 1] == 0

// Action 1: No action
#define ACTION_1  
// Action 2: New label (the block has foreground pixels and is not connected to anything else)
#define ACTION_2  
//Action P: Merge with block P
#define ACTION_3 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols - 2); 
// Action Q: Merge with block Q
#define ACTION_4 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols);	
// Action R: Merge with block R
#define ACTION_5 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols + 2); 
// Action S: Merge with block S
#define ACTION_6 Union(b_labels, b_labels_index, b_labels_index - 2);  
// Action 7: Merge labels of block P and Q
#define ACTION_7 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols - 2); Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols);			
//Action 8: Merge labels of block P and R
#define ACTION_8 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols - 2); Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols + 2);			
// Action 9 Merge labels of block P and S
#define ACTION_9 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols - 2); Union(b_labels, b_labels_index, b_labels_index - 2);			
// Action 10 Merge labels of block Q and R
#define ACTION_10 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols); Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols + 2);			
// Action 11: Merge labels of block Q and S
#define ACTION_11 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols); Union(b_labels, b_labels_index, b_labels_index - 2);			
// Action 12: Merge labels of block R and S
#define ACTION_12 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols + 2); Union(b_labels, b_labels_index, b_labels_index - 2);			
// Action 13: not used
#define ACTION_13 
// Action 14: Merge labels of block P, Q and S
#define ACTION_14 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols - 2); Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols); Union(b_labels, b_labels_index, b_labels_index - 2);		
//Action 15: Merge labels of block P, R and S
#define ACTION_15 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols - 2); Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols + 2); Union(b_labels, b_labels_index, b_labels_index - 2);			
//Action 16: labels of block Q, R and S
#define ACTION_16 Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols); Union(b_labels, b_labels_index, b_labels_index - 2 * label_cols + 2); Union(b_labels, b_labels_index, b_labels_index - 2);
            
            #include "drag_bg.inc.cuh"

#undef ACTION_1
#undef ACTION_2
#undef ACTION_3
#undef ACTION_4
#undef ACTION_5
#undef ACTION_6
#undef ACTION_7
#undef ACTION_8
#undef ACTION_9
#undef ACTION_10
#undef ACTION_11
#undef ACTION_12
#undef ACTION_13
#undef ACTION_14
#undef ACTION_15
#undef ACTION_16


#undef CONDITION_B
#undef CONDITION_C
#undef CONDITION_D
#undef CONDITION_E

#undef CONDITION_G
#undef CONDITION_H
#undef CONDITION_I
#undef CONDITION_J
#undef CONDITION_K

#undef CONDITION_M
#undef CONDITION_N
#undef CONDITION_O
#undef CONDITION_P

#undef CONDITION_R
#undef CONDITION_S
#undef CONDITION_T
    }
}

__global__ void Path_Compression(unsigned int* labels, unsigned int* b_labels, const int img_rows, const int img_cols) {
    const int label_rows = img_rows;
    const int label_cols = img_cols;
    unsigned int row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    unsigned int col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    unsigned int labels_index = row * label_cols + col;
    unsigned int b_labels_index = row * label_cols + col;

    if (row < label_rows && col < label_cols) {
        labels[labels_index] = Find(labels, labels_index);
        b_labels[labels_index] = Find(b_labels, b_labels_index);
    }
}

__global__ void Distribute_Labels(unsigned int* img, unsigned int* labels, unsigned int* b_labels, const int img_rows, const int img_cols) {
    const int label_rows = img_rows;
    const int label_cols = img_cols;
    unsigned row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    unsigned col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    unsigned img_index = row * img_cols + col;
    unsigned labels_index = row * label_cols + col;

    if (row < label_rows && col < label_cols) {
        unsigned int background_label = b_labels[labels_index] + (unsigned int)(label_rows * label_cols);
        unsigned int foreground_label = labels[labels_index];
        labels[labels_index] = img[img_index]==0 ? background_label: foreground_label;

        if (col + 1 < label_cols) {
            labels[labels_index + 1] = img[img_index + 1]==0 ? background_label: foreground_label;
        }
        if (row + 1 < label_rows) {
            labels[labels_index + label_cols] = img[img_index + img_cols]==0 ? background_label: foreground_label;
        }
        if (col + 1 < label_cols && row + 1 < label_rows) {
            labels[labels_index + label_cols + 1] = img[img_index + img_cols + 1]==0 ? background_label: foreground_label;
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << "<input_file> <output_file> <test times>\n";
        exit(EXIT_FAILURE);
    }

    const char* input_filename = argv[1];
    const char* output_filename = argv[2];
    int test_times = atoi(argv[3]);

    FILE* input_file = fopen(input_filename, "r");
    if (input_file == NULL) {
        perror("Error opening input file");
        exit(EXIT_FAILURE);
    }

    int rows, cols; 
    if(!fscanf(input_file, "%d %d", &rows, &cols))
    {
        std::cerr << "Error reading file: size\n";
        exit(EXIT_FAILURE);
    }

    unsigned int* img = (unsigned int*)malloc(rows * cols * sizeof(int));
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if(!fscanf(input_file, "%d", &img[i * cols + j]))
            {
                std::cerr << "Error reading file: pixels\n";
                exit(EXIT_FAILURE);
            }
        }
    }

    unsigned int* labels = (unsigned int*)malloc(rows * cols * sizeof(unsigned int));
    
    unsigned int* d_img;
    unsigned int* d_labels;
    unsigned int* db_labels;
    hipMalloc(&d_img, rows * cols * sizeof(int));
    hipMalloc(&d_labels, rows * cols * sizeof(int));
    hipMalloc(&db_labels, rows * cols * sizeof(int));
    
    std::vector<double> running_times;
    dim3 grid_size_ = dim3((((cols + 1) / 2) + BLOCK_COLS - 1) / BLOCK_COLS, (((rows + 1) / 2) + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
    dim3 block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);
    for (int i = 0; i < test_times; i++) {
        hipDeviceSynchronize();
        auto start_time = std::chrono::high_resolution_clock::now();
        hipMemcpy(d_img, img, rows * cols * sizeof(unsigned int), hipMemcpyHostToDevice);
        Init_Labeling<<<grid_size_, block_size_>>>(d_labels, db_labels, rows, cols);
        Merge<<<grid_size_, block_size_>>>(d_img, d_labels, db_labels, rows, cols);
        Path_Compression<<<grid_size_, block_size_>>>(d_labels, db_labels, rows, cols);
        Distribute_Labels<<<grid_size_, block_size_>>>(d_img, d_labels, db_labels, rows, cols);
        hipMemcpy(labels, d_labels, rows * cols * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> running_time = end_time - start_time;
        running_times.push_back(running_time.count());
    }

    FILE* fp = fopen(output_filename, "w");
    if (fp == NULL) {
        std::cerr << "Error opening file";
        exit(EXIT_FAILURE);
    }

    // First line: print the number of rows and columns
    fprintf(fp, "%d %d\n", rows, cols);

    // Starting from the second line, print the result of the Union Find
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(fp, "%d ", labels[i * cols + j]);
        }
        fprintf(fp, "\n");
    }

    fclose(fp);
    fclose(input_file);

    std::cout << "CUDA_CCL Average Time: " << std::accumulate(running_times.begin(), running_times.end(), 0.0) / running_times.size() << "s" << std::endl;

    free(img);
    free(labels);
    hipFree(d_img);
    hipFree(d_labels);
    hipFree(db_labels);

}